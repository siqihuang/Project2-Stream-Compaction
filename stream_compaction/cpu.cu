#include <cstdio>
#include "cpu.h"
#include "common.h"
//#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace StreamCompaction {
namespace CPU {

/**
 * CPU scan (prefix sum).
 */
void scan(int n, int *odata, const int *idata) {
    // TODO
    if(n==0) return ;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	odata[0]=0;
	for(int i=1;i<n;++i){
		odata[i]=odata[i-1]+idata[i-1];
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout<<"Time used in scan on CPU "<<milliseconds<<" ms"<<std::endl;
}

/**
 * CPU stream compaction without using the scan function.
 *
 * @returns the number of elements remaining after compaction.
 */
int compactWithoutScan(int n, int *odata, const int *idata) {
    // TODO
    int count=0;
	for(int i=0;i<n;++i){
		if(idata[i]!=0) odata[count++]=1;
	}
	return count;
}

/**
 * CPU stream compaction using scan and scatter, like the parallel version.
 *
 * @returns the number of elements remaining after compaction.
 */
int compactWithScan(int n, int *odata, const int *idata) {
    // TODO
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	int *tmp1=new int[n],*tmp2=new int[n];
	for(int i=0;i<n;++i){
		if(idata[i]==0) tmp1[i]=0;
		else tmp1[i]=1;
	}
    scan(n,tmp2,tmp1);
	for(int i=0;i<n;++i){
		if(tmp1[i]!=0){
			odata[tmp2[i]]=1;
		}
	}
	int tmp=tmp2[n-1];
	delete tmp1;
	delete tmp2;

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout<<"Time used in compaction on CPU "<<milliseconds<<" ms"<<std::endl;
	
	return tmp;
}

}
}

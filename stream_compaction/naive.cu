#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
namespace Naive {

// TODO: __global__

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
__global__ void scanOnGPU(int n, int *odata, int *idata,int step){
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	if(index<n){
		if(index>=step) odata[index]=idata[index]+idata[index-step];
		else odata[index]=idata[index];
	}
}

void scan(int n, int *odata, const int *idata) {
    // TODO
    int step=1,count=0;
	int *dev_odata,*dev_idata;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipMalloc((void**)&dev_odata, n * sizeof(int));
	hipMalloc((void**)&dev_idata, n * sizeof(int));
	hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_odata, idata, n * sizeof(int), hipMemcpyHostToDevice);

	dim3 blockPerGrid=((n+blockSize-1)/blockSize);
	hipEventRecord(start);
	while(step<n){
		if(count%2==0) scanOnGPU<<<blockPerGrid,blockSize>>>(n,dev_odata,dev_idata,step);
		else scanOnGPU<<<blockPerGrid,blockSize>>>(n,dev_idata,dev_odata,step);
		count++;
		step*=2;
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	if(count%2==1) hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
	else hipMemcpy(odata, dev_idata, n*sizeof(int), hipMemcpyDeviceToHost);
	for(int i=n-1;i>0;--i){
		odata[i]=odata[i-1];
	}
	odata[0]=0;
	hipFree(dev_odata);
	hipFree(dev_idata);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout<<"Time used in naive scan on GPU "<<milliseconds<<" ms"<<std::endl;
}

}
}

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include <iostream>
namespace StreamCompaction {
namespace Efficient {

// TODO: __global__
__global__ void upSwapOnGPU(int *idata,int step,int n,int newN){
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	if(index<newN){
		if(step==1&&index>=n) idata[index]=0;
		if((index+1)%(step*2)==0) idata[index]+=idata[index-step];
	}
}

__global__ void downSwapOnGPU(int *idata,int step,int n,int newN){
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	if(index<newN){
		if(step*2==newN&&index==newN-1) idata[index]=0;
		if((index+1)%(step*2)==0){
			int tmp=idata[index-step];
			idata[index-step]=idata[index];
			idata[index]+=tmp;
		}
	}
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
    // TODO
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	int newN=pow(2,ilog2ceil(n));
	int *dev_idata;
	hipMalloc((void**)&dev_idata, newN * sizeof(int));
	hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

	dim3 blockPerGrid=((newN+blockSize-1)/blockSize);
	
	int step=1;
	hipEventRecord(start);
	while(step<newN){
		upSwapOnGPU<<<blockPerGrid,blockSize>>>(dev_idata,step,n,newN);
		step*=2;
	}
	step/=2;
	while(step!=0){
		downSwapOnGPU<<<blockPerGrid,blockSize>>>(dev_idata,step,n,newN);
		step/=2;
	}
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout<<"Time used in efficient scan on GPU "<<milliseconds<<" ms"<<std::endl;

	hipMemcpy(odata,dev_idata,n*sizeof(int),hipMemcpyDeviceToHost);
	hipFree(dev_idata);
}

__global__ void countOne(int n,int *idata,int *odata){
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	if(index<n){
		if(idata[index]!=0) odata[index]=1;
		else odata[index]=0;
	}
}

__global__ void getCompact(int *idata,int *tmp,int *odata,int n){
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	if(index<n){
		if(idata[index]==1) odata[tmp[index]]=idata[index];
	}
}
/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata) {
    // TODO
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    int *dev_tmp1,*dev_tmp2,*dev_idata,*dev_odata,*tmp1=new int[n],*tmp2=new int[n];
	hipMalloc((void**)&dev_tmp1,n*sizeof(int));
	hipMalloc((void**)&dev_tmp2,n*sizeof(int));
	hipMalloc((void**)&dev_idata,n*sizeof(int));
	hipMalloc((void**)&dev_odata,n*sizeof(int));
	hipMemcpy(dev_idata,idata,n*sizeof(int),hipMemcpyHostToDevice);

	dim3 blockPerGrid=((n+blockSize-1)/blockSize);
	hipEventRecord(start);
	countOne<<<blockPerGrid,blockSize>>>(n,dev_idata,dev_tmp1);
	hipMemcpy(tmp1,dev_tmp1,n*sizeof(int),hipMemcpyDeviceToHost);
	scan(n,tmp2,tmp1);
	hipMemcpy(dev_tmp2,tmp2,n*sizeof(int),hipMemcpyHostToDevice);
	getCompact<<<blockPerGrid,blockSize>>>(dev_tmp1,dev_tmp2,dev_odata,n);
	hipMemcpy(odata,dev_odata,n*sizeof(int),hipMemcpyDeviceToHost);
	int count=tmp2[n-1]+tmp1[n-1];

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	delete tmp1;
	delete tmp2;
	hipFree(dev_tmp1);
	hipFree(dev_tmp2);
	hipFree(dev_idata);
	hipFree(dev_odata);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout<<"Time used in compaction on GPU "<<milliseconds<<" ms"<<std::endl;

	return count;
}

}
}

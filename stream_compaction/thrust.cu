#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
namespace Thrust {

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
    // TODO use `thrust::exclusive_scan`
    // example: for device_vectors dv_in and dv_out:
    // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
	std::vector<int> in;
	for(int i=0;i<n;++i) in.push_back(idata[i]);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	thrust::device_vector<int> dv_in(in.begin(),in.end());
	thrust::device_vector<int> dv_out(n,0);
	thrust::exclusive_scan(dv_in.begin(),dv_in.end(),dv_out.begin());
	std::vector<int> out(dv_out.begin(),dv_out.end());
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout<<"Time used in thrust scan on GPU "<<milliseconds<<" ms"<<std::endl;
	for(int i=0;i<n;++i){
		odata[i]=out[i];
	}
}

}
}

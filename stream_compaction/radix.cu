#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "radix.h"
#include "efficient.h"
#include "common.h";
#include <iostream>

namespace StreamCompaction {
namespace Radix {

void scan(int n, int *odata, const int *idata){
	StreamCompaction::Efficient::scan(n,odata,idata);
}

__device__ int getDigit(int n,int pos){
	int result=0;
	for(int i=0;i<pos;++i){
		result=n%2;
		n/=2;
	}
	return result;
}

__global__ void getDigits(int n, int *idata, int *odata,int pos){
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	if(index<n){
		odata[index]=getDigit(idata[index],pos);
	}
}

__global__ void Reverse(int n,int *idata,int *odata){
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	if(index<n){
		odata[index]=1-idata[index];
	}
}

__global__ void getT(int n,int *idata,int *odata,int totalFalse){
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	if(index<n){
		odata[index]=index-idata[index]+totalFalse;
	}
}

__global__ void getPos(int n,int *dev_b,int *dev_t,int *dev_f,int *odata){
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	if(index<n){
		odata[index]=dev_b[index]*dev_t[index]+(1-dev_b[index])*dev_f[index];
	}
}

__global__ void switchPos(int n,int *idata, int *odata, int *dev_d){
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	if(index<n){
		odata[dev_d[index]]=idata[index];
	}
}

void radix(int n, int *odata, const int *idata){
	int num=ilog2ceil(n-1);
	int *dev_idata,*dev_odata,*dev_b,*dev_e,*dev_f,*dev_t,*dev_d;
	int *host_f=new int[n],*host_e=new int[n];
	hipMalloc((void**)&dev_idata, n*sizeof(int));
	hipMalloc((void**)&dev_odata, n*sizeof(int));
	hipMalloc((void**)&dev_b, n*sizeof(int));
	hipMalloc((void**)&dev_e, n*sizeof(int));
	hipMalloc((void**)&dev_f, n*sizeof(int));
	hipMalloc((void**)&dev_t, n*sizeof(int));
	hipMalloc((void**)&dev_d, n*sizeof(int));
	hipMemcpy(dev_idata,idata,n*sizeof(int),hipMemcpyHostToDevice);

	dim3 blockPerGrid((n+blockSize-1)/blockSize);
	for(int i=1;i<=num;++i){
		if(i%2==1) getDigits<<<blockPerGrid,blockSize>>>(n,dev_idata,dev_b,i);
		else getDigits<<<blockPerGrid,blockSize>>>(n,dev_odata,dev_b,i);
		Reverse<<<blockPerGrid,blockSize>>>(n,dev_b,dev_e);
		hipMemcpy(host_e,dev_e,n*sizeof(int),hipMemcpyDeviceToHost);
		scan(n,host_f,host_e);
		hipMemcpy(dev_f,host_f,n*sizeof(int),hipMemcpyHostToDevice);
		int totalFalse=host_e[n-1]+host_f[n-1];
		getT<<<blockPerGrid,blockSize>>>(n,dev_f,dev_t,totalFalse);
		getPos<<<blockPerGrid,blockSize>>>(n,dev_b,dev_t,dev_f,dev_d);
		if(i%2==1) switchPos<<<blockPerGrid,blockSize>>>(n,dev_idata,dev_odata,dev_d);
		else switchPos<<<blockPerGrid,blockSize>>>(n,dev_odata,dev_idata,dev_d);
	}

	if(num%2==1) hipMemcpy(odata,dev_odata,n*sizeof(int),hipMemcpyDeviceToHost);
	else hipMemcpy(odata,dev_idata,n*sizeof(int),hipMemcpyDeviceToHost);

	hipFree(dev_idata);
	hipFree(dev_odata);
	hipFree(dev_b);
	hipFree(dev_t);
	hipFree(dev_d);
	hipFree(dev_f);
	hipFree(dev_e);
}

}
}
